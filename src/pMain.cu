#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <iostream>
#include <cstdlib>
#include <vector>
#include <time.h>
#include "pNetwork/loaderMnist.h"
#include "pNetwork/pNetwork.h"
#include "common/utilsTime.h"
//#include "sNetwork/Network.h"



int main()
{
	system("script pLog.txt");
	//Constructor MnistLoader
	MnistLoader mnist = MnistLoader("../MNIST/train-images.idx3-ubyte", 
									"../MNIST/t10k-images.idx3-ubyte", 
									"../MNIST/train-labels.idx1-ubyte", 
									"../MNIST/t10k-labels.idx1-ubyte");
	
	//METODO PARA IMPRIMIR TODOS LOS DATOS: 0 IMPRIME DATOS DE ENTRENAMIENTO, 
	//										1 IMPRIME DATOS DE PRUEBA
	//mnist.print_data_set(0, 10);	
	//cout << mnist.train_data.size() << endl;
	
	//CREO LA ESTRUCTURA DE LA RED
	vector<int> sizes;	
	sizes.push_back(784); sizes.push_back(100); sizes.push_back(15); sizes.push_back(10);			
	//sizes.push_back(3); sizes.push_back(5); sizes.push_back(2);;			
	Network_P net = Network_P(sizes);	
	//Network net = Network(sizes);	
	//net.mostrar_pesos();

	
	//Network_P net1 = Network_P(sizes);	

	net.load("pNet");

	//net1.mostrar_pesos();
	//net.mostrar_pesos(); 
	//printf("Error: %s \n", hipGetErrorName(hipGetLastError()));
	//net.test_network(mnist.train_data, 50);
	int EPOCAS = 6000;
	double ERROR = 0.000001;
	double RATELEARNING = 0.07;	

	time_t first, second;
	first = time(NULL);  
	//tStart = clock();
	vector<double> erroes = net.train_backpropagation(mnist.test_data, RATELEARNING, EPOCAS, ERROR, 10);		
	//tEnd = clock();
	second = time(NULL);
	cout << "Tiempo entrenamiento: " << difftime(second, first) << " segundos\n";

	//clock_t train_time = tEnd-tStart;
	//cout << "Tiempo de entrenamiento en red paralela: " << (train_time/CLOCKS_PER_SEC)/(60*60*24) << endl;
	tm train_time = getTm(difftime(second, first));
	cout << "tm_struct: " << train_time.tm_hour << ":" << train_time.tm_min << ":" << train_time.tm_sec << endl;
	net.test_network(mnist.train_data, 10);
	net.save("pNet");
	//net.mostrar_output();*/
	net.test_network(mnist.test_data, 10);
	system("exit");
    return 0;
}
