#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <iostream>
#include <cstdlib>
#include <vector>
#include <time.h>
#include "common/loaderMnist.h"
#include "pNetwork/pNetwork.h"
#include "common/utilsTime.h"

int main()
{
	//Constructor MnistLoader
	MnistLoader mnist = MnistLoader("../MNIST/data/train-images.idx3-ubyte", 
									"../MNIST/data/tdata/10k-images.idx3-ubyte", 
									"../MNIST/data/train-labels.idx1-ubyte", 
									"../MNIST/data/t10k-labels.idx1-ubyte");

	int typeNet = 0; // 0 PARALLEL, 1 SEQUENTIAL
	vector<int> sizes;	
	sizes.push_back(784); sizes.push_back(100); sizes.push_back(10);			
	if(typeNet == 0)
	{
		Network_P net = Network_P(sizes);		
	}
	else
	{
		Network net = Network(sizes);		
	}	
	int EPOCAS = 500;
	double ERROR = 0.003;
	double RATELEARNING = 0.5;
	int ExTrain = 60000;
	int ExTest = 10000;		

	
	cout << "Datos de la Red" << endl;
	cout << "Estructura: ";
	for (int i = 0; i < sizes.size(); ++i)
	{
		cout << sizes[i] << ", ";
	}
	cout << endl;
	cout << "Velocidad de Aprendizaje: " << RATELEARNING << endl;
	cout << "Error min: " << ERROR << endl;
	cout << "Tipo Algoritmo: "Secuencial << endl;
	cout << "Ejemplos en entrenamiento: " << ExTrain	<< "	Ejemplos de prueba: " << ExTest << endl;
	cout << "Cantidad de epocas: " << EPOCAS << endl;

	vector<double> errores;
	printTime(errores = net.train_backpropagation(mnist.train_data, RATELEARNING, EPOCAS, ERROR, 60000));		
	//printTime(net.test_network(mnist.train_data, 10000));
	printTime(net.test_network(mnist.test_data, 10000));
	cout << "Error promedio alcanzado: " << errores[errores.size()-1] << endl;
	net.save("../models/pNet_0");
    return 0;
}
