#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <iostream>
#include <cstdlib>
#include <vector>
#include <time.h>
#include "pNetwork/loaderMnist.h"
#include "pNetwork/pNetwork.h"

int main()
{
	clock_t tStart, tEnd;
	//Constructor MnistLoader
	MnistLoader mnist = MnistLoader("../MNIST/train-images.idx3-ubyte", 
									"../MNIST/t10k-images.idx3-ubyte", 
									"../MNIST/train-labels.idx1-ubyte", 
									"../MNIST/t10k-labels.idx1-ubyte");
	
	//METODO PARA IMPRIMIR TODOS LOS DATOS: 0 IMPRIME DATOS DE ENTRENAMIENTO, 
	//										1 IMPRIME DATOS DE PRUEBA
	//mnist.print_data_set(0, 10);	
	//cout << mnist.train_data.size() << endl;
	
	//CREO LA ESTRUCTURA DE LA RED
	vector<int> sizes;	
	sizes.push_back(784); sizes.push_back(100); sizes.push_back(50); sizes.push_back(10);			
	//sizes.push_back(3); sizes.push_back(5); sizes.push_back(2);;			
	Network_P net = Network_P(sizes);	
	//net.mostrar_pesos();

	
	//Network_P net1 = Network_P(sizes);	

	//net1.load("pNet");

	//net1.mostrar_pesos();
	//net.mostrar_pesos(); 
	//printf("Error: %s \n", hipGetErrorName(hipGetLastError()));
	//net.test_network(mnist.train_data, 50);
	int EPOCAS = 6000;
	double ERROR = 0.0001;
	double RATELEARNING = 10;	
	
	tStart = clock();
	net.train_backpropagation(mnist.train_data, RATELEARNING, EPOCAS, ERROR, 600);		
	tEnd = clock();
	clock_t train_time = tEnd-tStart;
	cout << "Tiempo de entrenamiento en red paralela: " << (train_time/CLOCKS_PER_SEC)/(60*60*24) << endl;
	//net.test_network(mnist.test_data, 100);
	net.test_network(mnist.train_data, 1000);
	net.save("pNet");
	//net.mostrar_output();*/
    return 0;
}
