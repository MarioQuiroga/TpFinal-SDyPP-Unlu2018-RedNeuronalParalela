#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <iostream>
#include <cstdlib>
#include <vector>
#include <time.h>
#include "common/loaderMnist.h"
#include "pNetwork/pNetwork.h"
#include "sNetwork/Network.h"
#include "common/utilsTime.h"

int main()
{
	//Constructor MnistLoader
	MnistLoader mnist = MnistLoader("../MNIST/data/train-images.idx3-ubyte", 
									"../MNIST/data/t10k-images.idx3-ubyte", 
									"../MNIST/data/train-labels.idx1-ubyte", 
									"../MNIST/data/t10k-labels.idx1-ubyte", 784, 10);

	int typeNet = 1; // 0 PARALLEL, 1 SEQUENTIAL
	vector<int> sizes;	
	sizes.push_back(784); sizes.push_back(500); sizes.push_back(300); sizes.push_back(100); sizes.push_back(100);  sizes.push_back(10);			
	int EPOCAS = 10;
	double ERROR = 0.003;
	double RATELEARNING = 0.5;
	int ExTrain = 60000;
	int ExTest = 10000;		
	cout << "Datos de la Red" << endl;
	cout << "Estructura: ";
	for (int i = 0; i < sizes.size(); ++i)
	{
		cout << sizes[i] << ", ";
	}
	cout << endl;
	cout << "Velocidad de Aprendizaje: " << RATELEARNING << endl;
	cout << "Error min: " << ERROR << endl;
	cout << "Tipo Algoritmo: "<< typeNet << endl;
	cout << "Ejemplos en entrenamiento: " << ExTrain	<< "	Ejemplos de prueba: " << ExTest << endl;
	cout << "Cantidad de epocas: " << EPOCAS << endl;
	vector<double> errores;
	if(typeNet == 0)
	{
		Network_P net = Network_P(sizes);		
		printTime(errores = net.train_backpropagation(mnist.train_data, RATELEARNING, EPOCAS, ERROR, ExTrain));		
		printTime(net.test_network(mnist.test_data, ExTest));
		cout << "Error promedio alcanzado: " << errores[errores.size()-1] << endl;
		net.save("../models/pNet_3");
	}
	else
	{
		Network net = Network(sizes);		
		printTime(errores = net.train_backpropagation(mnist.train_data, RATELEARNING, EPOCAS, ERROR, ExTrain));		
		printTime(net.test_network(mnist.test_data, ExTest));
		cout << "Error promedio alcanzado: " << errores[errores.size()-1] << endl;
		net.save("../models/sNet_3");
	}	
    return 0;
}
