#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <iostream>
#include <cstdlib>
#include <vector>
#include <time.h>
#include "common/loaderMnist.h"
#include "pNetwork/pNetwork.h"
#include "common/utilsTime.h"

int main()
{
	//Constructor MnistLoader
	MnistLoader mnist = MnistLoader("../MNIST/train-images.idx3-ubyte", 
									"../MNIST/t10k-images.idx3-ubyte", 
									"../MNIST/train-labels.idx1-ubyte", 
									"../MNIST/t10k-labels.idx1-ubyte");
	
	//METODO PARA IMPRIMIR TODOS LOS DATOS: 0 IMPRIME DATOS DE ENTRENAMIENTO, 
	//										1 IMPRIME DATOS DE PRUEBA
	//mnist.print_data_set(0, 10);	
	//cout << mnist.train_data.size() << endl;
	
	//CREO LA ESTRUCTURA DE LA RED
	vector<int> sizes;	
	sizes.push_back(784); sizes.push_back(100); sizes.push_back(15); sizes.push_back(10);			
	//sizes.push_back(3); sizes.push_back(5); sizes.push_back(2);;			
	Network_P net = Network_P(sizes);	
	//Network net = Network(sizes);	
	//net.mostrar_pesos();
	
	//Network_P net1 = Network_P(sizes);	

	//net.load("../models/pNet1_2");

	//net1.mostrar_pesos();
	//net.mostrar_pesos(); 
	//printf("Error: %s \n", hipGetErrorName(hipGetLastError()));
	//net.test_network(mnist.train_data, 50);
	int EPOCAS = 1000;
	double ERROR = 0.003;
	double RATELEARNING = 0.5;	
	cout << "RATELEARNING: " << RATELEARNING << endl;
	
	printTime(vector<double> erroes = net.train_backpropagation(mnist.train_data, RATELEARNING, EPOCAS, ERROR, 60000));		
	printTime(net.test_network(mnist.train_data, 10000));
	printTime(net.test_network(mnist.test_data, 10000));
	net.save("../models/pNet3");
    return 0;
}
