#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <iostream>
#include <cstdlib>
#include <vector>
#include <time.h>
#include "common/loaderMnist.h"
#include "pNetwork/pNetwork.h"

int main()
{
	clock_t tStart, tEnd;
	//Constructor MnistLoader
	MnistLoader mnist = MnistLoader("../MNIST/train-images.idx3-ubyte", 
									"../MNIST/t10k-images.idx3-ubyte", 
									"../MNIST/train-labels.idx1-ubyte", 
									"../MNIST/t10k-labels.idx1-ubyte");
	
	//METODO PARA IMPRIMIR TODOS LOS DATOS: 0 IMPRIME DATOS DE ENTRENAMIENTO, 
	//										1 IMPRIME DATOS DE PRUEBA
	//mnist.print_data_set(0);	
	//cout << mnist.train_data.size() << endl;
	
	//CREO LA ESTRUCTURA DE LA RED
	vector<int> sizes;	
	sizes.push_back(784); sizes.push_back(100); sizes.push_back(50); sizes.push_back(15); sizes.push_back(10);			
	Network_P net = Network_P(sizes);	
	
	//net.mostrar_pesos();  
	//net.test_network(mnist.train_data, 50);
	int EPOCAS = 6000;
	double ERROR = 0.00000001;
	double RATELEARNING = 0.5;		
	
	tStart = clock();
	net.train_backpropagation(mnist.train_data, RATELEARNING, EPOCAS, ERROR, 600);		
	tEnd = clock();
	clock_t train_time = tEnd-tStart;
	cout << "Tiempo de entrenamiento en red paralela: " << train_time << endl;
	//net.test_network(mnist.test_data, 10000);
	//net.mostrar_output();

    return 0;
}
